#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

//A = imagen original
//B = imagen resultante
//N = dimensiones (NxN) Matriz Cuadrada
//V = nivel de vecindad
//Bs = tamaño de bloque

__global__ void suma2D(){

}

void randomImage(float *A, int size){
    for(int i = 0; i < size; i++){
        A[i] = (float)rand()/RAND_MAX;
    }
}

void printImage(float *A, int size, int N){
    int j = 0;
    for(int i = 0; i < size; i++){
        printf("%f ",A[i]);
        j++;
        if(j == N){
            printf("\n");
            j = 0;
        }

    }
}

int main(void){
    //Variables
    //TODO: GetOpt
    int N = 5;
    int size = N*N;
    int V = 1;

    //Pedir memoria en host
    float *h_A = (float *)malloc(size*sizeof(float));
    
    //Generación de imagen random
    randomImage(h_A, size);
    printImage(h_A, size, N);

    dim3 blockSize = dim3(N/Bs, N/Bs);
    dim3 gridSize = dim3(Bs,Bs);

    suma2D<<gridSize,blockSize>>(A,B,N,V);

    
}


/*COSAS POR HACER
- suma2D
- suma2Dshmem
- suma2D_CPU
- No sé si hay que agregar __host__ a las funciones que no son __global__
*/