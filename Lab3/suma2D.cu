#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

//A = imagen original
//B = imagen resultante
//N = dimensiones (NxN) Matriz Cuadrada
//V = nivel de vecindad
//Bs = tamaño de bloque

__global__ void suma2D(float *A, float *B, int N, int V){

}

void randomImage(float *A, int N){
    for(int i = 0; i < N*N; i++){
        A[i] = (float)rand()/RAND_MAX;
    }
}

void printImage(float *A, int N){
    int j = 0;
    for(int i = 0; i < N*N; i++){
        printf("%f ",A[i]);
        j++;
        if(j == N){
            printf("\n");
            j = 0;
        }

    }
}

int main(void){
    //Variables
    //TODO: GetOpt
    int N = 5;
    int size = N*N;
    int V = 1;
    int Bs = 1;

    //Pedir memoria en host
    float *h_A = (float *)malloc(size*sizeof(float));
    float *h_B = (float *)malloc(size*sizeof(float));

    //Generación de imagen random
    randomImage(h_A, N);
    printImage(h_A, N);

    //Pedir memoria en device
    float *d_A, *d_B;
    hipMalloc((void **) &d_A, size*sizeof(float));
    hipMalloc((void **) &d_B, size*sizeof(float));

    //Copia desde Host a Device
    hipMemcpy(d_A, h_A, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size*sizeof(float), hipMemcpyHostToDevice);

    //Llamado a la función de suma
    dim3 blockSize = dim3(N/Bs, N/Bs);
    dim3 gridSize = dim3(Bs,Bs);
    suma2D<<gridSize,blockSize>>(d_A, d_B, N, V);

    //Copia desde Device a Host
    hipMemcpy(&h_B, d_B, size*sizeof(float), hipMemcpyDeviceToHost);
    printImage(h_B, N);
    
}


/*COSAS POR HACER
- suma2D
- suma2Dshmem
- suma2D_CPU
- No sé si hay que agregar __host__ a las funciones que no son __global__
*/