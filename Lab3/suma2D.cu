
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <ctype.h>
#include <string.h>
#include <time.h> 

//Ejemplo compilacion: nvcc suma2D.cu -o suma2D
//Ejemplo ejecucion: ./suma2D -N 5 -B 1 -V 1


//A = imagen original
//B = imagen resultante
//N = dimensiones (NxN) Matriz Cuadrada
//V = nivel de vecindad
//Bs = tamaño de bloque

__global__ void suma2D(float *A, float *B, int N, int V){
    int i, j;
    i = blockDim.x * blockIdx.x + threadIdx.x; //horizontal
    j = blockDim.y * blockIdx.y + threadIdx.y; //vertical

    B[i * N + j] = 0.0;

    for (int a = i-V; a <= i+V; a++){
        for (int b = j-V; b <= j+V; b++){
            if(a >= 0 && a < N && b >= 0 && b < N){
                B[i * N + j] = B[i * N + j] + A[a * N + b];
            }
        }
    }
}

void suma2D_CPU(float *A, float *B, int N, int V){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            B[i * N + j] = 0.0;
            for (int a = i-V; a <= i+V; a++){
                for (int b = j-V; b <= j+V; b++){
                    if(a >= 0 && a < N && b >= 0 && b < N){ 
                        B[i * N + j] = B[i * N + j] + A[a * N + b];
                    }
                }
            }
        }
    }
}

__host__ void randomImage(float *A, int N){
    for(int i = 0; i < N*N; i++){
        A[i] = (float)rand()/RAND_MAX;
        //A[i] = 1;
    }
}

__host__ void printImage(float *A, int N){
    int j = 0;
    for(int i = 0; i < N*N; i++){
        printf("%f\n", A[i]);
        j++;
    }
}

__host__ int main(int argc, char **argv){
    
    int N = 0;
    int Bs = 0;
    int V = 0;
    int c1;
    
    while((c1 =  getopt(argc, argv,"N:B:V:")) != -1){
        switch(c1){
            case 'N':
                if(atof(optarg) < 1){
                    printf("El valor ingresado debe ser mayor que 0\n");
                    c1 = -1;    
                    break;
                }
                else N = atof(optarg);
                break;
            case 'B':
                if(atof(optarg) < 1){
                    printf("El valor ingresado debe ser mayor que 0\n");
                    c1 = -1;    
                    break;
                }
                else Bs = atof(optarg);
                break;  
            case 'V':
                if(atof(optarg) < 1){
                    printf("El valor ingresado debe ser mayor que 0\n");
                    c1 = -1;    
                    break;
                }
                else V = atof(optarg);
                break;  
            case '?':
                if(optopt == 'N' || optopt == 'B' || optopt == 'V')
                    fprintf(stderr, "Option -%c requeries an argument.\n",optopt);
                else if(isprint(optopt))
                    fprintf(stderr,"Unknown option -%c.\n",optopt);
                else
                    fprintf(stderr, "Unknown option character `\\x%x'.\n",optopt);
                return 1;
            default:
                abort();
        }
    }

    int size = N*N;

    //Pedir memoria en host
    float *h_A = (float *)malloc(size*sizeof(float));
    float *h_B = (float *)malloc(size*sizeof(float));
    float *h_C = (float *)malloc(size*sizeof(float));
    //Generación de imagen random
    randomImage(h_A, N);
    printf("Imagen Original:\n ");
    printImage(h_A, N);
    printf("\n");

    //Se empieza a medir el tiempo en GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    //Pedir memoria en device
    float *d_A, *d_B;
    hipMalloc((void **) &d_A, size*sizeof(float));
    hipMalloc((void **) &d_B, size*sizeof(float));

    //Copia desde Host a Device
    hipMemcpy(d_A, h_A, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size*sizeof(float), hipMemcpyHostToDevice);

    //Llamado a la función de suma en GPU
    dim3 blockSize = dim3(Bs, Bs);
    dim3 gridSize = dim3(N/Bs,N/Bs);
    suma2D<<<gridSize,blockSize>>>(d_A, d_B, N, V);
    
    //Copia desde Device a Host
    hipMemcpy(h_B, d_B, size*sizeof(float), hipMemcpyDeviceToHost);

    //Se termine de medir el tiempo en GPU
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop);

    //Se imprime por consola la imagen nueva y el tiempo de ejecución en GPU
    printf("Imagen Resultante en GPU:\n ");
    printImage(h_B, N);
    printf("\n");

    //Se empieza a medir tiempo en CPU
    double time_spent = 0.0;
    clock_t begin = clock(); 

    //Llamado a la función de suma en CPU
    suma2D_CPU(h_A, h_C, N, V);
    printf("Imagen Resultante en CPU:\n ");
    printImage(h_C, N);

    //Se termina de medir tiempo en CPU
    clock_t end = clock(); 
    time_spent += (double)(end-begin)/CLOCKS_PER_SEC;

    printf("Tiempo de Ejecucion GPU: %f seg.\n", elapsedTime/1000);
    printf("Tiempo de Ejecucion CPU: %f seg.\n", time_spent);

    //Se libera memoria solicitada
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);

    exit(0);
    return 0;
}

