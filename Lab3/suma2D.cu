
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//A = imagen original
//B = imagen resultante
//N = dimensiones (NxN) Matriz Cuadrada
//V = nivel de vecindad
//Bs = tamaño de bloque

__global__ void suma2D(float *A, float *B, int N, int V){
    int i, j;
    i = blockDim.x * blockIdx.x + threadIdx.x; //horizontal
    j = blockDim.y * blockIdx.y + threadIdx.y; //vertical

    B[i * N + j] = 0.0;

    for (int a = i-V; a <= i+V; a++){
        for (int b = j-V; b <= j+V; b++){
            if(a >= 0 && a < N && b >= 0 && b < N){
                B[i * N + j] = B[i * N + j] + A[a * N + b];
            }
        }
    }
}

void suma2D_CPU(float *A, float *B, int N, int V){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            B[i * N + j] = 0.0;
            for (int a = i-V; a <= i+V; a++){
                for (int b = j-V; b <= j+V; b++){
                    if(a >= 0 && a < N && b >= 0 && b < N){ 
                        B[i * N + j] = B[i * N + j] + A[a * N + b];
                    }
                }
            }
        }
    }
}

__host__ void randomImage(float *A, int N){
    for(int i = 0; i < N*N; i++){
        A[i] = (float)rand()/RAND_MAX;
        //A[i] = 1;
    }
}

__host__ void printImage(float *A, int N){
    int j = 0;
    for(int i = 0; i < N*N; i++){
        printf("%f ", A[i]);
        j++;
        if(j == N){
            printf("\n");
            j = 0;
        }
    }
}

__host__ int main(void){
    //Variables
    //TODO: GetOpt
    int N = 5;
    int size = N*N;
    int V = 1;
    int Bs = 1;

    //Pedir memoria en host
    float *h_A = (float *)malloc(size*sizeof(float));
    float *h_B = (float *)malloc(size*sizeof(float));

    //Generación de imagen random
    randomImage(h_A, N);
    printImage(h_A, N);

    //Pedir memoria en device
    float *d_A, *d_B;
    hipMalloc((void **) &d_A, size*sizeof(float));
    hipMalloc((void **) &d_B, size*sizeof(float));

    //Copia desde Host a Device
    hipMemcpy(d_A, h_A, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size*sizeof(float), hipMemcpyHostToDevice);
    printf("\n\n");

    //Llamado a la función de suma en GPU
    dim3 blockSize = dim3(N/Bs, N/Bs);
    dim3 gridSize = dim3(Bs,Bs);
    suma2D<<<gridSize,blockSize>>>(d_A, d_B, N, V);
    
    //Copia desde Device a Host
    hipMemcpy(h_B, d_B, size*sizeof(float), hipMemcpyDeviceToHost);
    printImage(h_B, N);
     
    printf("\n\n");
    //Llamado a la función de suma en CPU
    suma2D_CPU(h_A, h_B, N, V);
    printImage(h_B, N);
    exit(0);
}


/*COSAS POR HACER
- suma2Dshme
*/

