
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//A = imagen original
//B = imagen resultante
//N = dimensiones (NxN) Matriz Cuadrada
//V = nivel de vecindad
//Bs = tamaño de bloque

__global__ void suma2D_SHMEM(float *A, float *B, int N, int V){
    extern __shared__ float data[];

    int i, j;
    i = blockDim.x * blockIdx.x + threadIdx.x; //horizontal
    j = blockDim.y * blockIdx.y + threadIdx.y; //vertical
    
    int tid = i * N + j;

    data[tid] = 0.0;

    for (int a = i-V; a <= i+V; a++){
        for (int b = j-V; b <= j+V; b++){
            if(a >= 0 && a < N && b >= 0 && b < N){
                data[i * N + j] = data[i * N + j] + A[a * N + b];
            }
        }
    }

    __syncthreads();
    B[tid] = data[tid];
}

void suma2D_CPU(float *A, float *B, int N, int V){
    //TODO: Quitar el elemento i,j dentro de la suma o preguntar
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            B[i * N + j] = 0.0;
            for (int a = i-V; a <= i+V; a++){
                for (int b = j-V; b <= j+V; b++){
                    if(a >= 0 && a < N && b >= 0 && b < N){ 
                        B[i * N + j] = B[i * N + j] + A[a * N + b];
                    }
                }
            }
        }
    }
}

void randomImage(float *A, int N){
    for(int i = 0; i < N*N; i++){
        A[i] = (float)rand()/RAND_MAX;
    }
}

void printImage(float *A, int N){
    int j = 0;
    for(int i = 0; i < N*N; i++){
        printf("%f ",A[i]);
        j++;
        if(j == N){
            printf("\n");
            j = 0;
        }

    }
}

__host__ int main(void){
    //Variables
    //TODO: GetOpt
    int N = 5;
    int size = N*N;
    int V = 1;
    int Bs = 1;

    //Pedir memoria en host
    float *h_A = (float *)malloc(size*sizeof(float));
    float *h_B = (float *)malloc(size*sizeof(float));

    //Generación de imagen random
    randomImage(h_A, N);
    printImage(h_A, N);

    //Pedir memoria en device
    float *d_A, *d_B;
    hipMalloc((void **) &d_A, size*sizeof(float));
    hipMalloc((void **) &d_B, size*sizeof(float));

    //Copia desde Host a Device
    hipMemcpy(d_A, h_A, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size*sizeof(float), hipMemcpyHostToDevice);

    //Llamado a la función de suma en GPU
    dim3 blockSize = dim3(N/Bs, N/Bs);
    dim3 gridSize = dim3(Bs,Bs);
    suma2D_SHMEM<<<gridSize,blockSize, size>>>(d_A, d_B, N, V);

    //Copia desde Device a Host
    hipMemcpy(h_B, d_B, size*sizeof(float), hipMemcpyDeviceToHost);
    printImage(h_B, N);

    //Llamado a la función de suma en CPU
    suma2D_CPU(h_A, h_B, N, V);

    exit(0);

}