//
//  Autor: Joaquín Ignacio Jara Marín - Shalini Henna Ramchandani Moorjimal
//  Rut: 19.739.353-k - 19.307.417-0
//  Curso: HPC
//  Profesor: Fernando Rannou
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <ctype.h>
#include <string.h>
#include <time.h>

//Funcion que genera una nueva imagen a partir de una vecindad indicada utilizando memoria compartida
//Entrada:  -Puntero de float que representa la imagen original
//          -Puntero de float que representa la imagen resultante
//          -N, Entero que indica el largo de una matriz de NxN
//          -V, Entero que indica el tamaño de la vecindad
__global__ void suma2D_SHMEM(float *A, float *B, int N, int V){
    __shared__ float data[1024];
    
    int i, j, k;
    i = blockDim.x * blockIdx.x + threadIdx.x;
    j = blockDim.y * blockIdx.y + threadIdx.y;
    k = i * N + j;
    
    data[i] = 0.0;

    for (int a = i-V; a <= i+V; a++){
        for (int b = j-V; b <= j+V; b++){
            if(a >= 0 && a < N && b >= 0 && b < N){
                data[i] = data[i] + A[a * N + b];
            }
        }
    }
    B[k] = data[i];
}

//Funcion que genera una nueva imagen a partir de una vecindad indicada en CPU
//Entrada:  -Puntero de float que representa la imagen original
//          -Puntero de float que representa la imagen resultante
//          -N, Entero que indica el largo de una matriz de NxN
//          -V, Entero que indica el tamaño de la vecindad
void suma2D_CPU(float *A, float *B, int N, int V){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            B[i * N + j] = 0.0;
            for (int a = i-V; a <= i+V; a++){
                for (int b = j-V; b <= j+V; b++){
                    if(a >= 0 && a < N && b >= 0 && b < N){ 
                        B[i * N + j] = B[i * N + j] + A[a * N + b];
                    }
                }
            }
        }
    }
}

//Funcion que genera una nueva imagen aleatoria de NxN
//Entrada:  -Puntero de float que genera una imagen con pixeles random
//          -N, Entero que indica el largo de una imagen de NxN
void randomImage(float *A, int N){
    for(int i = 0; i < N*N; i++){
        A[i] = (float)rand()/RAND_MAX;
    }
}

//Funcion que imprime los valores de una imagen
//Entrada:  -Puntero de float que representa una imagen
//          -N, Entero que indica el largo de una imagen de NxN
void printImage(float *A, int N){
    int j = 0;
    for(int i = 0; i < N*N; i++){
        printf("%f\n",A[i]);
        j++;
    }
}

int main(int argc, char **argv){
    
    int N = 0;
    int Bs = 0;
    int V = 0;
    int c1;
    
    while((c1 =  getopt(argc, argv,"N:B:V:")) != -1){
        switch(c1){
            case 'N':
                if(atof(optarg) < 1){
                    printf("El valor ingresado debe ser mayor que 0\n");
                    c1 = -1;    
                    break;
                }
                else N = atof(optarg);
                break;
            case 'B':
                if(atof(optarg) < 1){
                    printf("El valor ingresado debe ser mayor que 0\n");
                    c1 = -1;    
                    break;
                }
                else Bs = atof(optarg);
                break;  
            case 'V':
                if(atof(optarg) < 1){
                    printf("El valor ingresado debe ser mayor que 0\n");
                    c1 = -1;    
                    break;
                }
                else V = atof(optarg);
                break;  
            case '?':
                if(optopt == 'N' || optopt == 'B' || optopt == 'V')
                    fprintf(stderr, "Option -%c requeries an argument.\n",optopt);
                else if(isprint(optopt))
                    fprintf(stderr,"Unknown option -%c.\n",optopt);
                else
                    fprintf(stderr, "Unknown option character `\\x%x'.\n",optopt);
                return 1;
            default:
                abort();
        }
    }


    int size = N*N;

    //Pedir memoria en host
    float *h_A = (float *)malloc(size*sizeof(float));
    float *h_B = (float *)malloc(size*sizeof(float));
    float *h_C = (float *)malloc(size*sizeof(float));

    //Generación de imagen random
    randomImage(h_A, N);
    printf("Imagen Original:\n ");
    printImage(h_A, N);
    printf("\n");

    //Se empieza a medir el tiempo en GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    //Pedir memoria en device
    float *d_A, *d_B;
    hipMalloc((void **) &d_A, size*sizeof(float));
    hipMalloc((void **) &d_B, size*sizeof(float));

    //Copia desde Host a Device
    hipMemcpy(d_A, h_A, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size*sizeof(float), hipMemcpyHostToDevice);

    //Llamado a la función de suma en GPU
    dim3 blockSize = dim3(Bs, Bs);
    dim3 gridSize = dim3(N/Bs,N/Bs);
    suma2D_SHMEM<<<gridSize, blockSize>>>(d_A, d_B, N, V);

    //Copia desde Device a Host
    hipMemcpy(h_B, d_B, size*sizeof(float), hipMemcpyDeviceToHost);

    //Se termine de medir el tiempo en GPU
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop);

    //Se imprime por consola la imagen nueva y el tiempo de ejecución en GPU
    printf("Imagen Resultante en GPU:\n ");
    printImage(h_B, N);

    //Se empieza a medir tiempo en CPU
    double time_spent = 0.0;
    clock_t begin = clock(); 

    //Llamado a la función de suma en CPU
    suma2D_CPU(h_A, h_C, N, V);
    printf("Imagen Resultante en CPU:\n ");
    printImage(h_C, N);

    //Se termina de medir tiempo en CPU
    clock_t end = clock(); 
    time_spent += (double)(end-begin)/CLOCKS_PER_SEC;

    printf("Tiempo de Ejecucion GPU: %f seg.\n", elapsedTime/1000);
    printf("Tiempo de Ejecucion CPU: %f seg.\n", time_spent);
    
    //Se libera memoria solicitada
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);
    
    exit(0);

}