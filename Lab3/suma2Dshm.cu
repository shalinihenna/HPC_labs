
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <ctype.h>
#include <string.h>


//Ejemplo compilacion: nvcc suma2Dshm.cu -o suma2Dshm
//Ejemplo ejecucion: ./suma2Dshm -N 5 -B 1 -V 1

//A = imagen original
//B = imagen resultante
//N = dimensiones (NxN) Matriz Cuadrada
//V = nivel de vecindad
//Bs = tamaño de bloque

__global__ void suma2D_SHMEM(float *A, float *B, int N, int V){
    extern __shared__ float data[];

    int i, j;
    i = blockDim.x * blockIdx.x + threadIdx.x; //horizontal
    j = blockDim.y * blockIdx.y + threadIdx.y; //vertical
    
    int tid = i * N + j;

    data[tid] = 0.0;

    for (int a = i-V; a <= i+V; a++){
        for (int b = j-V; b <= j+V; b++){
            if(a >= 0 && a < N && b >= 0 && b < N){
                data[i * N + j] = data[i * N + j] + A[a * N + b];
            }
        }
    }

    __syncthreads();
    B[tid] = data[tid];
}

void suma2D_CPU(float *A, float *B, int N, int V){
    //TODO: Quitar el elemento i,j dentro de la suma o preguntar
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            B[i * N + j] = 0.0;
            for (int a = i-V; a <= i+V; a++){
                for (int b = j-V; b <= j+V; b++){
                    if(a >= 0 && a < N && b >= 0 && b < N){ 
                        B[i * N + j] = B[i * N + j] + A[a * N + b];
                    }
                }
            }
        }
    }
}

void randomImage(float *A, int N){
    for(int i = 0; i < N*N; i++){
        A[i] = (float)rand()/RAND_MAX;
        //A[i] = 1;
    }
}

void printImage(float *A, int N){
    int j = 0;
    for(int i = 0; i < N*N; i++){
        printf("%f ",A[i]);
        j++;
        if(j == N){
            printf("\n");
            j = 0;
        }

    }
}

__host__ int main(int argc, char **argv){
    
    int N = 0;
    int Bs = 0;
    int V = 0;
    int c1;
    
    while((c1 =  getopt(argc, argv,"N:B:V:")) != -1){
        switch(c1){
            case 'N':
                if(atof(optarg) < 1){
                    printf("El valor ingresado debe ser mayor que 0\n");
                    c1 = -1;    
                    break;
                }
                else N = atof(optarg);
                break;
            case 'B':
                if(atof(optarg) < 1){
                    printf("El valor ingresado debe ser mayor que 0\n");
                    c1 = -1;    
                    break;
                }
                else Bs = atof(optarg);
                break;  
            case 'V':
                if(atof(optarg) < 1){
                    printf("El valor ingresado debe ser mayor que 0\n");
                    c1 = -1;    
                    break;
                }
                else V = atof(optarg);
                break;  
            case '?':
                if(optopt == 'N' || optopt == 'B' || optopt == 'V')
                    fprintf(stderr, "Option -%c requeries an argument.\n",optopt);
                else if(isprint(optopt))
                    fprintf(stderr,"Unknown option -%c.\n",optopt);
                else
                    fprintf(stderr, "Unknown option character `\\x%x'.\n",optopt);
                return 1;
            default:
                abort();
        }
    }

    int size = N*N;

    //Pedir memoria en host
    float *h_A = (float *)malloc(size*sizeof(float));
    float *h_B = (float *)malloc(size*sizeof(float));

    //Generación de imagen random
    randomImage(h_A, N);
    printf("\n\nImagen original\n");
    printImage(h_A, N);

    //Pedir memoria en device
    float *d_A, *d_B;
    hipMalloc((void **) &d_A, size*sizeof(float));
    hipMalloc((void **) &d_B, size*sizeof(float));

    //Copia desde Host a Device
    hipMemcpy(d_A, h_A, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size*sizeof(float), hipMemcpyHostToDevice);

    //Llamado a la función de suma en GPU
    dim3 blockSize = dim3(N/Bs, N/Bs);
    dim3 gridSize = dim3(Bs,Bs);
    suma2D_SHMEM<<<gridSize, blockSize, size>>>(d_A, d_B, N, V);

    //Copia desde Device a Host
    hipMemcpy(h_B, d_B, size*sizeof(float), hipMemcpyDeviceToHost);
    printf("\n\nImagen sumada desde GPU con memoria compartida\n");
    printImage(h_B, N);

    //Llamado a la función de suma en CPU
    suma2D_CPU(h_A, h_B, N, V);
    printf("\n\nImagen sumada desde CPU\n");
    printImage(h_B, N);
    exit(0);

}