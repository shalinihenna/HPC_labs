
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h> 

//A = imagen original
//B = imagen resultante
//N = dimensiones (NxN) Matriz Cuadrada
//V = nivel de vecindad
//Bs = tamaño de bloque

__global__ void suma2D_SHMEM(float *A, float *B, int N, int V){
    extern __shared__ float data[];

    int i, j;
    i = blockDim.x * blockIdx.x + threadIdx.x; //horizontal
    j = blockDim.y * blockIdx.y + threadIdx.y; //vertical
    
    int tid = i * N + j;

    data[tid] = 0.0;

    for (int a = i-V; a <= i+V; a++){
        for (int b = j-V; b <= j+V; b++){
            if(a >= 0 && a < N && b >= 0 && b < N){
                data[i * N + j] = data[i * N + j] + A[a * N + b];
            }
        }
    }

    __syncthreads();
    B[tid] = data[tid];
}

void suma2D_CPU(float *A, float *B, int N, int V){
    //TODO: Quitar el elemento i,j dentro de la suma o preguntar
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            B[i * N + j] = 0.0;
            for (int a = i-V; a <= i+V; a++){
                for (int b = j-V; b <= j+V; b++){
                    if(a >= 0 && a < N && b >= 0 && b < N){ 
                        B[i * N + j] = B[i * N + j] + A[a * N + b];
                    }
                }
            }
        }
    }
}

void randomImage(float *A, int N){
    for(int i = 0; i < N*N; i++){
        A[i] = (float)rand()/RAND_MAX;
    }
}

void printImage(float *A, int N){
    int j = 0;
    for(int i = 0; i < N*N; i++){
        printf("%f ",A[i]);
        j++;
        if(j == N){
            printf("\n");
            j = 0;
        }

    }
}

__host__ int main(void){
    //Variables
    //TODO: GetOpt
    int N = 5;
    int size = N*N;
    int V = 1;
    int Bs = 1;

    //Pedir memoria en host
    float *h_A = (float *)malloc(size*sizeof(float));
    float *h_B = (float *)malloc(size*sizeof(float));

    //Generación de imagen random
    randomImage(h_A, N);
    printf("Imagen Original:\n ");
    printImage(h_A, N);
    printf("\n\n");

    //Se empieza a medir el tiempo en GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    //Pedir memoria en device
    float *d_A, *d_B;
    hipMalloc((void **) &d_A, size*sizeof(float));
    hipMalloc((void **) &d_B, size*sizeof(float));

    //Copia desde Host a Device
    hipMemcpy(d_A, h_A, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size*sizeof(float), hipMemcpyHostToDevice);

    //Llamado a la función de suma en GPU
    dim3 blockSize = dim3(N/Bs, N/Bs);
    dim3 gridSize = dim3(Bs,Bs);
    suma2D_SHMEM<<<gridSize,blockSize, size>>>(d_A, d_B, N, V);

    //Copia desde Device a Host
    hipMemcpy(h_B, d_B, size*sizeof(float), hipMemcpyDeviceToHost);

    //Se termine de medir el tiempo en GPU
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop);

    //Se imprime por consola la imagen nueva y el tiempo de ejecución en GPU
    printf("Imagen Resultante en GPU:\n ");
    printImage(h_B, N);
    printf("Tiempo de Ejecucion GPU: %3.lf ms.\n", elapsedTime);
    printf("\n\n");

    //Se empieza a medir tiempo en CPU
    double time_spent = 0.0;
    clock_t begin = clock(); 

    //Llamado a la función de suma en CPU
    suma2D_CPU(h_A, h_B, N, V);
    printf("Imagen Resultante en CPU:\n ");
    printImage(h_B, N);

    //Se termina de medir tiempo en CPU
    clock_t end = clock(); 
    time_spent += (double)(end-begin)/CLOCKS_PER_SEC;
    printf("Tiempo de Ejecucion CPU: %f seg.\n", time_spent);

    //Se libera memoria solicitada
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);

    exit(0);

}