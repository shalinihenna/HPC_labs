
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024*1024
#define T 128


__global__ void dotproduct(float *a, float *b, float *result)
{
	int i = threadIdx.x;
	int j = blockIdx.x*T + i;
	__shared__ float temp[T];

	temp[i] = a[j] * b[j];

	__syncthreads();

	if (threadIdx.x == 0) {
		float sum = 0.0;
		for (i=0; i < T; i++) sum += temp[i];
	
		atomicAdd(result, sum);
	}
}


int main() {

	float *a = (float *) malloc(N*sizeof(float));
	float *b = (float *) malloc(N*sizeof(float));
	float c = 0.0;
	int i;

	for (i=0; i < N; i++)
		a[i] = b[i] = 1.0;

	float *d_a, *d_b, *d_c;

	hipMalloc((void **) &d_a, N*sizeof(float));
	hipMalloc((void **) &d_b, N*sizeof(float));
	hipMalloc((void **) &d_c, sizeof(float));
	hipMemcpy(d_a, a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, &c, sizeof(float), hipMemcpyHostToDevice);

	dotproduct<<<N/T, T>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, sizeof(float), hipMemcpyDeviceToHost);

	printf("c = %f\n", c);

	exit(0);
}

